
#include <hip/hip_runtime.h>
#include <cstdio>
__global__ void addArrays(int *a, int *b, int *c, int size) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    printf("index:\n%d", index);
    c[index] = a[index] + b[index];
  }
}

int main() {
  int size = 1000;
  size_t bytes = size * sizeof(int);
  int *h_a = (int *)malloc(bytes);
  int *h_b = (int *)malloc(bytes);
  int *h_c = (int *)malloc(bytes);

  for (int i = 0; i < size; ++i) {
    h_a[i] = i;
    h_b[i] = 2 * i;
  }

  int *d_a, *d_b, *d_c;
  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);

  hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

  dim3 blockSize(256, 1, 1);
  dim3 gridSize((size + blockSize.x - 1) / blockSize.x, 1, 1);

  addArrays<<<gridSize, blockSize>>>(d_a, d_b, d_c, size);
  hipDeviceSynchronize();

  hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

  // for (int i = 0; i < size; ++i) {
  //   printf("%d + %d = %d\n", h_a[i], h_b[i], h_c[i]);
  // }

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  free(h_a);
  free(h_b);
  free(h_c);

  return 0;
}
